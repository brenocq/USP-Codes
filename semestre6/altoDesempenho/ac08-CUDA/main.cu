#include "hip/hip_runtime.h"
// SSC0903 - Computação de Alto Desempenho
// AC08 - Multiplicacao de matrizes quadradas com CUDA
//
// Breno Cunha Queiroz
// Matheus Branco Borella
// Natan Bernardi Cerdeira

#include <stdio.h>
#include <stdlib.h>

__global__ 
void multiFast(double *A, double* B, double *C, int N, int blockSize)
{
    extern __shared__ double shd[];
    double* dataA = shd;
    double* dataB = shd + N*blockSize;

    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    //----- Load -----//
    int numToLoad = N/blockSize;

    // Load A
    int startCol = numToLoad*threadIdx.x;
    int endCol = startCol + numToLoad;
    if(threadIdx.x == blockSize-1)
        endCol = N;
    for(unsigned i = startCol; i < endCol; i++)
        dataA[threadIdx.y*N+i] = A[row*N + i];

    // Load B
    startCol = numToLoad*threadIdx.y;
    endCol = startCol + numToLoad;
    if(threadIdx.y == blockSize-1)
        endCol = N;
    for(unsigned i = startCol; i < endCol; i++)
        dataB[threadIdx.x*N+i] = B[col*N + i];
    __syncthreads();

    //----- Calculate partial result -----//
    if(row < N && col < N)
    {
        double dot = 0;
        for(unsigned i = 0; i < N; i++)
            dot += dataA[threadIdx.y*N + i] * dataB[threadIdx.x*N + i];
        C[row*N + col] = dot;
    }
}

__global__ 
void multiSlow(double *A, double* B, double *C, int N, int blockSize)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    //----- Calculate partial result -----//
    if(row < N && col < N)
    {
        double dot = 0;
        for(unsigned i = 0; i < N; i++)
            dot += A[row*N + i] * B[col*N + i];
        C[row*N + col] = dot;
    }
}

int main(int argc, char **argv)
{
    // Declare matrices
    double *mA_h, *mBt_h, *mC_h, *mA_d, *mBt_d, *mC_d; 
    unsigned N, i, j; 

    // Read matrix dimensions
    fscanf(stdin,"%d\n",&N); 

    // Allocate matrices
    mA_h = (double*)malloc(N*N*sizeof(double));
    mBt_h = (double*)malloc(N*N*sizeof(double));// B transposed
    mC_h = (double*)malloc(N*N*sizeof(double));
    hipMalloc(&mA_d, N*N*sizeof(double));
    hipMalloc(&mBt_d, N*N*sizeof(double));
    hipMalloc(&mC_d, N*N*sizeof(double));

    // Read matrices
    for(i=0;i<N;i++)
        for(j=0;j<N;j++)
            fscanf(stdin, "%lf ", &mA_h[i*N + j]);
    for(i=0;i<N;i++)
        for(j=0;j<N;j++)
            fscanf(stdin, "%lf ", &mBt_h[j*N + i]);

    // Copy to GPU
    hipMemcpy(mA_d, mA_h, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(mBt_d, mBt_h, N*N*sizeof(double), hipMemcpyHostToDevice);

    // Calculate block size
    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    // Solve C=A*B
    // Calculate block size, should not be greater than 16x16 (256 threads)
    // If N is less than 16, use N
    // If maximum shared mememory per block is not enough, use maximum block size possible
    int blockSize = min(
            unsigned(min(16, N)),
            unsigned(prop.sharedMemPerBlock/(N*2*sizeof(double))));

    if(blockSize > 0)
    {
        // Multiply using shared memory to cache block values
        dim3 gridDim = dim3((N+blockSize-1)/blockSize, (N+blockSize-1)/blockSize, 1);
        dim3 blockDim = dim3(blockSize, blockSize, 1);
        multiFast<<<gridDim, blockDim, blockSize*N*2*sizeof(double)>>>(mA_d, mBt_d, mC_d, N, blockSize);
    }
    else
    {
        // Multiply without using shared memory if the matrix is too big
        dim3 gridDim = dim3((N+15)/16, (N+15)/16, 1);
        dim3 blockDim = dim3(16, 16, 1);
        multiSlow<<<gridDim, blockDim>>>(mA_d, mBt_d, mC_d, N, 16);
    }

    hipMemcpy(mC_h, mC_d, N*N*sizeof(double), hipMemcpyDeviceToHost);

    // Show result
    for(i=0;i<N;i++)
    {
        for(j=0;j<N;j++)
            printf("%.1lf ", mC_h[i*N+j]);
        printf("\n");
    }

    // Deallocate matrices
    free(mA_h);
    free(mBt_h);
    free(mC_h);
    hipFree(mA_d);
    hipFree(mBt_d);
    hipFree(mC_d);

    return 0;
}
