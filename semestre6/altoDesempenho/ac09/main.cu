// SSC0903 - Computação de Alto Desempenho
// AC09 - Valor minimo de matrizes quadradas com CUDA
//
// Breno Cunha Queiroz
// Matheus Branco Borella
// Natan Bernardi Cerdeira


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <limits.h>

// Kernel to find the minimum
__global__ void findMinimum(int *mat, unsigned L, unsigned C, int* res)
{
    // Threads on the same block find the minimum using the shared memory
    extern __shared__ int shd[];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Calculate how many value to minimize in this block
    int blockSize = blockDim.x;
    if(blockIdx.x == gridDim.x-1);
        blockSize = (L*C) % blockDim.x;
 
    if(idx < L*C)
    {
        // Load from global memory to shared memory;
        shd[threadIdx.x] = mat[idx];
        __syncthreads();

        // Similar to merge sort, every iteration reduce by half the number of elements to find the minimum
        for(int it = 1; blockSize/it > 0; it*=2)
        {
            if(threadIdx.x < ceil(blockSize/float(it)))
            {
                // Calculate partial minimum
                if(threadIdx.x*it*2 + it < blockSize)
                    shd[threadIdx.x*it*2] = min(shd[threadIdx.x*it*2], shd[threadIdx.x*it*2 + it]);
            }

            // Wait all threads finish their work
            __syncthreads();
        }

        // Thread 0 updates the global result
        if(threadIdx.x == 0)
            atomicMin(res, shd[0]);
    }
}


int main()
{
    // Declarations
    int *A_h, *B_h, *A_d, *B_d, *resA_d, *resB_d;
    int resA_h = INT_MAX, resB_h = INT_MAX;
    unsigned L1, C1, L2, C2, i, j; 
    hipStream_t stream1, stream2;

    // Read matrix dimensions
    fscanf(stdin,"%d %d\n%d %d", &L1, &C1, &L2, &C2); 

    // Allocate matrices
    A_h = (int*)malloc(L1*C1*sizeof(int));
    B_h = (int*)malloc(L2*C2*sizeof(int));
    hipMalloc(&A_d, L2*C2*sizeof(int));
    hipMalloc(&B_d, L2*C2*sizeof(int));
    hipMalloc(&resA_d, sizeof(int));
    hipMalloc(&resB_d, sizeof(int));

    // Read matrices
    for(i = 0; i < L1; i++)
        for(j = 0; j < C1; j++)
            fscanf(stdin, "%d", &A_h[i*C1+j]);
    for(i = 0; i < L2; i++)
        for(j = 0; j < C2; j++)
            fscanf(stdin, "%d", &B_h[i*C2+j]);

    // Create streams
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    //----- Run kernels -----//
    // Copy from CPU to GPU
    hipMemcpyAsync(resA_d, &resA_h, sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(A_d, A_h, L1*C1*sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(resB_d, &resB_h, sizeof(int), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(B_d, B_h, L2*C2*sizeof(int), hipMemcpyHostToDevice, stream2);

    // Run kernels
    findMinimum<<<(L1*C1+511)/512, 512, 512*sizeof(int), stream1>>>(A_d, L1, C1, resA_d);
    findMinimum<<<(L2*C2+511)/512, 512, 512*sizeof(int), stream2>>>(B_d, L2, C2, resB_d);

    // Synchronize
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Copy result from GPU to CPU
    hipMemcpyAsync(&resA_h, resA_d, sizeof(int), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(&resB_h, resB_d, sizeof(int), hipMemcpyDeviceToHost, stream2);

    // Print result
    fprintf(stdout, "%d\n%d", resA_h, resB_h);

    // Deallocate
    free(A_h);
    free(B_h);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(resA_d);
    hipFree(resB_d);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}
